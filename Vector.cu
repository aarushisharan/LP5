//%%writefile vector.cu

#include <hip/hip_runtime.h>
#include<iostream>
#include<bits/stdc++.h>
#include<chrono>

using namespace std;
using namespace std::chrono;

__global__ void add(int *A, int *B, int *C, int size)
{
    int tid= blockIdx.x * blockDim.x + threadIdx.x;
    if(tid<size)
    {
        C[tid]=A[tid]+B[tid];
    }
}

void initialize(int* vec, int size)
{
    for(int i=0;i<size;i++)
    {
        cout<<"Enter element "<<i+1<<"of the vector: ";
        cin>>vec[i];
    }
}

void print(int* vec, int size)
{
    for(int i=0;i<size;i++)
    {
        cout<<vec[i]<<" ";

    }
    cout<<endl;

}

void Seq_Addition(int *A, int *B, int *C, int size)
{
    for(int i=0;i<size;i++)
    {
        C[i]=A[i]+B[i];
    }

}

int main()
{

    int N;
    cout<<"Enter the size of vectors= "<<endl;
    cin>>N;
    int* A, * B, * C;

    int vectorsize=N;
    size_t vectorBytes = vectorsize*sizeof(int);
    
    A = new int[vectorsize];
    B = new int[vectorsize];
    C = new int[vectorsize];

    initialize(A, N);
    initialize(B, N);

    cout << "Vector A: ";
    print(A, N);
    cout << "Vector B: ";
    print(B, N);

    int *X, *Y, *Z;
    hipMalloc(&X,vectorBytes);
    hipMalloc(&Y,vectorBytes);
    hipMalloc(&Z,vectorBytes);

    hipMemcpy(X,A,vectorBytes,hipMemcpyHostToDevice);
    hipMemcpy(Y,B,vectorBytes,hipMemcpyHostToDevice);

    int threadsPerBlock= 256;
    int blocksPerGrid= (N+threadsPerBlock-1)/threadsPerBlock;

    //Sequential Addition
    auto start = high_resolution_clock::now();
    Seq_Addition(A, B, C, N);
    auto stop = high_resolution_clock::now();
    auto seq_duration = duration_cast<microseconds>(stop - start);

    cout << "Sequential Addition: ";
    print(C, N);

    //Parallel Addition
    start=high_resolution_clock::now();
    add<<<blocksPerGrid,threadsPerBlock>>>(X,Y,Z,N);
    hipMemcpy(C,Z,vectorBytes, hipMemcpyDeviceToHost);
    stop=high_resolution_clock::now();
    auto par_duration = duration_cast<microseconds>(stop - start);

    cout << "Parallel Addition: ";
    print(C, N);

    cout << "Sequential Addition Time: " << seq_duration.count() << " microseconds" << endl;
    cout << "Parallel Addition Time: " << par_duration.count() << " microseconds" << endl;

    

    hipFree(X);
    hipFree(Y);
    hipFree(Z);
    return 0;
}

